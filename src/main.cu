#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include<GL/glew.h>
#include<iostream>
#include <GLFW/glfw3.h>
#include "Shader.h"
#include "VertexBuffer.h"
#include "IndexBuffer.h"
#include "vert_array_quick_bind.h"
#include "Global_Bind_Test.h"
#include "freecam.h"
#include <functional>

#include <fstream>
#include <sstream>
#include "artefact.h"
#include <stb_image/stb_image.h>
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#include "Texture.h"

#include "ray.cuh"







using namespace glm;
#define gpuCheckErrs(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
using namespace std;
freecam primary_cam;
void MouseControlWrapper(GLFWwindow* window, double mouse_x, double mouse_y) {
	primary_cam.mouse_handler(window, mouse_x, mouse_y);
}
void ScrollControlWrapper(GLFWwindow* window, double x_disp, double y_disp) {
	primary_cam.scroll_handler(window, x_disp, y_disp);
}




__device__ float sphere_ray_hit_test(const vec3 center, float radius,  ray r) {
	vec3 oc = r.get_origin() - center;
	float a = glm::dot(r.get_direction(), r.get_direction());  
	float h =  glm::dot(oc, r.get_direction());
	float c = glm::dot(oc, oc) - radius * radius;
	float discriminant = h * h -  a * c;
	if (discriminant < 0) {
		return -1.0f;
	}
	else {
		return (-h - sqrt(discriminant)) / ( a);
	}
	
}


__device__ vec3 pix_data(ray r, unsigned char* sky, int su, int sv ) {
	float t = sphere_ray_hit_test(vec3(0.f, 0.f, -1.5f), 0.5f, r);
	if (t > 0.f)
	{
		vec3 N = r.get_point_at_t(t) - vec3(0.f, 0.f, -1.5f);
		return 0.5f * vec3(N.x + 1, N.y + 1, N.z + 1);
		
	}
	else
	{
		
		vec3 sky_col;
		int index = sv * 1920 * 3 + su * 3;
		int r = (int)sky[index] ;
		float rc = (float)((float)r / 255);
		int g = (int)sky[index+1];
		float gc = (float)((float)g / 255);
		int b = (int)sky[index+2];
		float bc = (float)((float)b / 255);
		sky_col.x = rc;
		sky_col.y = gc;
		sky_col.z = bc;
		return sky_col;
	}

}


__global__ void render(unsigned char* pix_buff_loc, int max_x, int max_y, glm::vec3 lower_left_corner, glm::vec3 horizontal, glm::vec3 vertical, glm::vec3 origin, unsigned char*sky) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x * 4 + i * 4;
	auto u = float(i) / max_x;
	auto v = float(j) / max_y;
	ray r1(origin, lower_left_corner + u * horizontal + v * vertical);
	vec3 col = pix_data(r1, sky, i, j);
	unsigned char r = (int)(255 * col.x);
	unsigned char g = (int)(255 * col.y);
	unsigned char b = (int)(255 * col.z);
	pix_buff_loc[pixel_index + 0] = (int)r + 1;
	pix_buff_loc[pixel_index + 1] = (int)g ;
	pix_buff_loc[pixel_index + 2] = (int)b ;
	pix_buff_loc[pixel_index + 3] = 255;
}



__global__ void
paint(unsigned char* g_odata)
{
	int i = threadIdx.x;
	int off = i * 4;
	g_odata[off] = 180;  //red channel
	g_odata[off + 1] = 0;  //green channel
	g_odata[off + 2] = 255;   //blue channel
	g_odata[off + 3] = 255;   // alpha channel
}

int main()
{
	hipSetDevice(0);

	GLFWwindow* window;
	if (!glfwInit())
		return -1;
	window = glfwCreateWindow(1920, 1080, "CUDA project", glfwGetPrimaryMonitor(), NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glewInit();
	float vertices[] = {
		// positions          // colors           // texture coords
		1.f,  1.f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
		1.f, -1.f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
		-1.f, -1.f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
		-1.f,  1.f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left
	};
	unsigned int indices[] = {
		0, 1, 3,
		1, 2, 3
	};

	VertexBuffer vb(vertices, sizeof(vertices));
	IndexBuffer eb(indices, 6);
	VertexArray va;
	
	va.spec_vertex_size(8);
	va.add_layout_spec(3);
	va.add_layout_spec(3);
	va.add_layout_spec(2);
	va.AddBuffer(vb);
	Texture t;
	int width, height, nrChannels;
	width = 1920;
	height = 1080;
	nrChannels = 4;

	unsigned int pbo;
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * nrChannels * sizeof(GLubyte), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	hipGraphicsResource* res;
	gpuCheckErrs(hipGraphicsGLRegisterBuffer(&res, pbo, cudaGraphicsMapFlagsNone));
	gpuCheckErrs(hipGraphicsMapResources(1, &res, 0));
	unsigned char* out_data;
	size_t num_bytes;
	gpuCheckErrs(hipGraphicsResourceGetMappedPointer((void**)&out_data, &num_bytes, res));

	int tx = 8;//threads x
	int ty = 8;//threads y
	dim3 blocks(width / tx + 1, height / ty + 1);
	dim3 threads(tx, ty);

	int w, h, n;
	stbi_set_flip_vertically_on_load(true);
	unsigned char* data = stbi_load("res/textures/sky3.jpg", &w, &h, &n, 0);
	unsigned char* sky;
	hipMalloc(&sky, w * h * 3);
	hipMemcpy(sky, data, w * h * 3, hipMemcpyHostToDevice);


	vec3 lower_left_corner(-1.6, -0.9, -1.0);
	vec3 horizontal(3.2, 0.0, 0.0);
	vec3 vertical(0.0, 1.8, 0.0);
	vec3 origin(0.0, 0.0, 0.0);
	render << <blocks, threads >> > (out_data, width, height, lower_left_corner, horizontal, vertical, origin, sky);
	hipGraphicsUnmapResources(1, &res);

	
	



	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glUnmapBuffer(GL_PIXEL_UNPACK_BUFFER);
	t.use_pbo(width, height);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);



	Shader s("res/shaders/tex_basic.shader");
	glfwSetCursorPosCallback(window, MouseControlWrapper);
	glfwSetScrollCallback(window, ScrollControlWrapper);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glDisable(GL_CULL_FACE);

	while (!glfwWindowShouldClose(window))
	{
		primary_cam.input_handler(window);
		glClearColor(0.f, 0.f, 0.f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		s.Bind();
		t.Bind();
		va.Bind();
		eb.Bind();
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		va.Unbind();
		s.Unbind();
		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}
	glfwTerminate();
	return 0;
}