#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<GL/glew.h>
#include<iostream>
#include <GLFW/glfw3.h>
#include "Shader.h"
#include "VertexBuffer.h"
#include "IndexBuffer.h"
#include "vert_array_quick_bind.h"
#include "Global_Bind_Test.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include "freecam.h"
#include <functional>
#include <assimp/Importer.hpp>
#include <assimp/scene.h>
#include <assimp/postprocess.h>
#include <fstream>
#include <sstream>
#include "artefact.h"
#include <stb_image/stb_image.h>
#include "cudaGL.h"
#include "cuda_gl_interop.h"

#define gpuCheckErrs(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
using namespace std;
freecam primary_cam;
void MouseControlWrapper(GLFWwindow* window, double mouse_x, double mouse_y) {
	primary_cam.mouse_handler(window, mouse_x, mouse_y);
}
void ScrollControlWrapper(GLFWwindow* window, double x_disp, double y_disp) {
	primary_cam.scroll_handler(window, x_disp, y_disp);
}

__global__ void
paint(unsigned char* g_odata)
{
	int i = threadIdx.x;
	int off = i * 4;
	g_odata[off] = 0;
	g_odata[off + 1] = 0;
	g_odata[off + 2] = 255;
	g_odata[off + 3] = 255;
}

int main()
{
	hipSetDevice(0);

	GLFWwindow* window;
	if (!glfwInit())
		return -1;
	window = glfwCreateWindow(1920, 1080, "CUDA project", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glewInit();
	float vertices[] = {
		// positions          // colors           // texture coords
		1.f,  1.f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
		1.f, -1.f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
		-1.f, -1.f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
		-1.f,  1.f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left
	};
	unsigned int indices[] = {
		0, 1, 3,
		1, 2, 3
	};
	unsigned int VBO, VAO, EBO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	glGenBuffers(1, &EBO);
	glBindVertexArray(VAO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
	glEnableVertexAttribArray(2);
	unsigned int texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	int width, height, nrChannels;
	width = 5;
	height = 5;
	nrChannels = 4;

	unsigned int pbo;
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * nrChannels * sizeof(GLubyte), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	hipGraphicsResource* res;
	gpuCheckErrs(hipGraphicsGLRegisterBuffer(&res, pbo, cudaGraphicsMapFlagsNone));
	gpuCheckErrs(hipGraphicsMapResources(1, &res, 0));
	unsigned char* out_data;
	size_t num_bytes;
	gpuCheckErrs(hipGraphicsResourceGetMappedPointer((void**)&out_data, &num_bytes, res));
	paint << <1, 100 >> > (out_data);
	hipGraphicsUnmapResources(1, &res);
	unsigned char* h_in;
	h_in = (unsigned char*)malloc(width * height * nrChannels * sizeof(GLubyte));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glUnmapBuffer(GL_PIXEL_UNPACK_BUFFER);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glGenerateMipmap(GL_TEXTURE_2D);

	Shader s("res/shaders/tex_basic.shader");
	glfwSetCursorPosCallback(window, MouseControlWrapper);
	glfwSetScrollCallback(window, ScrollControlWrapper);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glDisable(GL_CULL_FACE);
	while (!glfwWindowShouldClose(window))
	{
		primary_cam.input_handler(window);

		glClearColor(0.f, 0.f, 0.f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		s.Bind();
		glBindTexture(GL_TEXTURE_2D, texture);
		glBindVertexArray(VAO);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		glBindVertexArray(0);
		s.Unbind();

		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}
	glfwTerminate();
	return 0;
}