#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<GL/glew.h>
#include<iostream>
#include <GLFW/glfw3.h>
#include "Shader.h"
#include "VertexBuffer.h"
#include "IndexBuffer.h"
#include "vert_array_quick_bind.h"
#include "Global_Bind_Test.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include "freecam.h"
#include <functional>
#include <assimp/Importer.hpp>
#include <assimp/scene.h>
#include <assimp/postprocess.h>
#include <fstream>
#include <sstream>
#include "artefact.h"
#include <stb_image/stb_image.h>
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#define TX 32
#define TY 32

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__global__
void distanceKernel(uchar4* d_out, int w, int h, int2 pos) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return; // Check if within image bounds
	const int i = c + r * w; // 1D indexing
	const int dist = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y));
	const unsigned char intensity = clip(255 - dist);

	d_out[i].x = 255;
	d_out[i].y = 0;
	d_out[i].z = 255;
	d_out[i].w = 255;
}

void kernelLauncher(uchar4* d_out, int w, int h, int2 pos) {
	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
	distanceKernel << <gridSize, blockSize >> > (d_out, w, h, pos);
}

#define gpuCheckErrs(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
using namespace std;
freecam primary_cam;
void MouseControlWrapper(GLFWwindow* window, double mouse_x, double mouse_y) {
	primary_cam.mouse_handler(window, mouse_x, mouse_y);
}
void ScrollControlWrapper(GLFWwindow* window, double x_disp, double y_disp) {
	primary_cam.scroll_handler(window, x_disp, y_disp);
}
__global__ void square(int* devin, int* devout)
{
	int i = threadIdx.x;
	devout[i] = devin[i] * devin[i];
}

int main()
{
	GLFWwindow* window;
	if (!glfwInit())
		return -1;
	window = glfwCreateWindow(1920, 1080, "CUDA project", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glewInit();
	float vertices[] = {
		// positions          // colors           // texture coords
		1.f,  1.f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
		1.f, -1.f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
		-1.f, -1.f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
		-1.f,  1.f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left
	};
	unsigned int indices[] = {
		0, 1, 3,
		1, 2, 3
	};
	unsigned int VBO, VAO, EBO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	glGenBuffers(1, &EBO);
	glBindVertexArray(VAO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
	glEnableVertexAttribArray(2);
	unsigned int texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	int width, height, nrChannels;
	width = 100;
	height = 100;
	nrChannels = 4;

	vector<unsigned char> data2;
	for (int i = 0; i < height; i++) {
		for (int j = 0;j < width;j++) {
			data2.push_back(255);
			data2.push_back(0);
			data2.push_back(255);
			data2.push_back(255);
		}
	}

	unsigned int pbo;
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * nrChannels * sizeof(GLubyte), NULL, GL_DYNAMIC_DRAW);
	void* mappedBuffer = glMapBuffer(GL_PIXEL_UNPACK_BUFFER, GL_WRITE_ONLY);
	//memcpy(mappedBuffer, &data2[0], width * height * nrChannels * sizeof(GLubyte));
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);


	hipGraphicsResource* res;
	gpuCheckErrs(hipGraphicsGLRegisterBuffer(&res, pbo, cudaGraphicsMapFlagsNone));
	gpuCheckErrs(hipGraphicsMapResources(1, &res, 0));
	uchar4* out_data;
	size_t num_bytes;
	gpuCheckErrs(hipGraphicsResourceGetMappedPointer((void**)&out_data, &num_bytes, res));
	int2 loc = { width / 2, height / 2 };
	kernelLauncher(out_data, width, height, loc);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glUnmapBuffer(GL_PIXEL_UNPACK_BUFFER);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glGenerateMipmap(GL_TEXTURE_2D);

	Shader s("res/shaders/tex_basic.shader");
	glfwSetCursorPosCallback(window, MouseControlWrapper);
	glfwSetScrollCallback(window, ScrollControlWrapper);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glDisable(GL_CULL_FACE);
	while (!glfwWindowShouldClose(window))
	{
		primary_cam.input_handler(window);

		glClearColor(0.f, 0.f, 0.f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		s.Bind();
		glBindTexture(GL_TEXTURE_2D, texture);
		glBindVertexArray(VAO);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		glBindVertexArray(0);
		s.Unbind();

		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}
	glfwTerminate();
	return 0;
}